#include "hip/hip_runtime.h"
// MP 4 Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include    <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this
#define ELEMENT_NUM_PER_BLOCK BLOCK_SIZE << 1
#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

int ceil(int a, int b){
    return (a + b - 1) / b;
}

__global__ void reduction(float * input, float * output, int len) {
    //@@ Load a segment of the input vector into shared memory
    //@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index

    __shared__ float shared[ELEMENT_NUM_PER_BLOCK];

    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int base_start = bx * ELEMENT_NUM_PER_BLOCK;

    if (base_start + tx * 2 < len) {
        shared[tx] = input[base_start + tx * 2];
    } else {
        shared[tx] = 0;
    }

    if (base_start + tx * 2 + 1 < len) {
        shared[tx] += input[base_start + tx * 2 + 1];
    } else {
        shared[tx] += 0;
    }
    __syncthreads();

    for (int stride = BLOCK_SIZE / 2; stride >=1; stride /=2) {
        if (tx < stride) {
            shared[tx] = shared[tx + stride];
        }
        __syncthreads();
    }

    if (tx == 0) {
        output[tx] = shared[0];
    }
}


int main(int argc, char ** argv) {
    int ii;
    wbArg_t args;
    float * hostInput; // The input 1D list
    float * hostOutput; // The output list
    float * deviceInput;
    float * deviceOutput;
    int numInputElements; // number of elements in the input list
    int numOutputElements; // number of elements in the output list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numInputElements);

    // each block output one elements
    numOutputElements = ceil(numInputElements, ELEMENT_NUM_PER_BLOCK);
    hostOutput = (float*) malloc(numOutputElements * sizeof(float));

    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numInputElements);
    wbLog(TRACE, "The number of output elements in the input is ", numOutputElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    hipMalloc((void **)&deviceInput, sizeof(float) * numInputElements);
    hipMalloc((void **)&deviceOutput, sizeof(float) * numOutputElements);

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput, hostInput, sizeof(float) * numInputElements, hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input memory to the GPU.");
    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(numOutputElements, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    reduction<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, sizeof(float) * numOutputElements, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    /********************************************************************
     * Reduce output vector on the host
     * NOTE: One could also perform the reduction of the output vector
     * recursively and support any size input. For simplicity, we do not
     * require that for this lab.
     ********************************************************************/
    for (ii = 1; ii < numOutputElements; ii++) {
        hostOutput[0] += hostOutput[ii];
    }

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    hipFree(deviceInput);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, 1);

    free(hostInput);
    free(hostOutput);

    return 0;
}
