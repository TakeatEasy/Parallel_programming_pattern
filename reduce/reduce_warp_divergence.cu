#include "hip/hip_runtime.h"
#include "generator.h"


#define BLOCK_SIZE 1024

int ceil(int a, int b){
    return int((a + b - 1) / b);
}

__global__ void reduce_warp_divergence(float * in, int len) {
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int did = threadIdx.x + blockDim.x * blockIdx.x;

    if (did < len) {
        sdata[tid] = in[did];
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride >= 1 ; stride /= 2) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }

        __syncthreads();
    }

    if (tid == 0) {
        in[blockIdx.x] = sdata[tid];
    }
}

int main(int argc, char ** argv) {

    int inputLength = 1024;
    float * hostInput;
    float * hostOutput;
    float * deviceInput;

    hostInput = new float[inputLength];
    hostOutput = new float[inputLength];

    generate_array(hostInput, inputLength);

    float res = 0.0;
    for (int i = 0; i < inputLength; i++) {
        res += hostInput[i];
    }
    std::cout << "The result for cpu is:" << res << std::endl;

    //@@ Allocate GPU memory here
    hipMalloc((void**) &deviceInput, sizeof(float) * inputLength);

    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput, hostInput, sizeof(float) * inputLength, hipMemcpyHostToDevice);
    
    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(ceil(inputLength, BLOCK_SIZE), 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);
    
    //@@ Launch the GPU Kernel here
    reduce_warp_divergence<<<DimGrid, DimBlock>>>(deviceInput, inputLength);

    hipDeviceSynchronize();
    
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceInput, sizeof(float) * inputLength, hipMemcpyDeviceToHost);

    //@@ Free the GPU memory here
    hipFree(deviceInput);

    std::cout << "The result for gpu is:" << hostOutput[0] << std::endl;
    std::cout << std::endl;

    free(hostInput);
    free(hostOutput);

    return 0;
}