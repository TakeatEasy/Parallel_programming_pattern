#include "hip/hip_runtime.h"
#include <wb.h>

#define BLOCK_SIZE 8
#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

int ceil(int a, int b) {
    return int((a + b - 1) / b);
}


__global__ void matrixMultiply(float * A, float * B, float * C,
			       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {

    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    float value = 0.0;
    if (row < numCRows && col < numCColumns) {
        for (int i = 0; i < numAColumns; i++) {
            value += A[numAColumns * row + i] * B[i * numBColumns + col];
        }
        C[row * numCColumns + col] = value;
    }
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    //@@ Allocate the hostC matrix
    hostC = (float *) malloc(sizeof(float) * numCRows * numCColumns);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
    wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    hipMalloc((void**) &deviceA, sizeof(float) * numAColumns * numARows);
    hipMalloc((void**) &deviceB, sizeof(float) * numBColumns * numBRows);
    hipMalloc((void**) &deviceC, sizeof(float) * numCColumns * numCRows);


    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    hipMemcpy(deviceA, hostA, sizeof(float) * numAColumns * numARows, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float) * numBColumns * numBRows, hipMemcpyHostToDevice);

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(ceil(numCColumns * numCRows, BLOCK_SIZE), ceil(numCColumns * numCRows, BLOCK_SIZE), 1);
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE,1);
    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostC, deviceC, sizeof(float) * numCColumns * numCRows, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    wbTime_stop(GPU, "Freeing GPU Memory");

    float res = 0;
    for(int index = 0; index < numAColumns; index ++){
        res += hostA[index] * hostB[index * numBColumns];
    }
    printf("res is \t %f\n", res);

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}