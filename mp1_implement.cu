#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>
#include <iostream>
#define BLOCK_SIZE 256

int ceil(int a, int b){
    return int((a + b - 1) / b);
}
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < len){
        out[idx] = in1[idx] + in2[idx];
    }
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

	wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    hipMalloc((void**) &deviceInput1, sizeof(float) * inputLength);
    hipMalloc((void**) &deviceInput2, sizeof(float) * inputLength);
    hipMalloc((void**) &deviceOutput, sizeof(float) * inputLength);

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput1, hostInput1, sizeof(float) * inputLength, hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, sizeof(float) * inputLength, hipMemcpyHostToDevice);


    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(ceil(inputLength, BLOCK_SIZE), 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, sizeof(float) * inputLength, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    //for(int index = 0; index < inputLength; index ++){
    //    printf("%f\t%f\t%f\n", hostInput1[index],hostInput2[index],hostOutput[index]);
    //}

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

