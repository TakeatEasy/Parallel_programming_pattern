#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include "generator.h"

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
void cudaCheck(hipError_t error) {
    if (error != hipSuccess) {
        exit(EXIT_FAILURE);
    }
    return;
};

void cpuSgemm(
    float *a, float *b, float *c, const int M, const int N, const int K) {
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = psum;
        }
    }
}

__global__ void naiveSgemm(
    float * a, float * b, float * c,
    const int M, const int N, const int K) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    if (m < M && n < N) {
        
        float psum = 0.0;
        #pragma unroll
        for (int k = 0; k < K; k++) {
            
            psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = psum;
    }
}

float testMaxError(
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a = new float[M * K];
    float *h_b = new float[K * N];
    float *h_c = new float[M * N];
    float *h_d_c = new float[M * N];
    float *d_a, *d_b, *d_c;

    generate_array(h_a, M * K);
    generate_array(h_b, K * N);
    generate_array(h_c, M * N);
    generate_array(h_d_c, M * N);

    cudaCheck(hipMalloc(&d_a, size_a));
    cudaCheck(hipMalloc(&d_b, size_b));
    cudaCheck(hipMalloc(&d_c, size_c));

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    
    cudaCheck(hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice));
    naiveSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    cudaCheck(hipDeviceSynchronize());

    cudaCheck(hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost));

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs(h_d_c[i] - h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}

int main() {

    printf("\nKernal = naiveSgemm\n");

    const int BM = 32, BN = 32;
    {
        const int M = 512, N = 512, K = 512;
        dim3 blockDim(BM, BN);
        dim3 gridDim((M + BM - 1) / BM, (N + BN - 1) / BN);
        float max_error = testMaxError(gridDim, blockDim, M, N, K);
        printf("Max Error = %f\n", max_error);
    }

    return 0;
}