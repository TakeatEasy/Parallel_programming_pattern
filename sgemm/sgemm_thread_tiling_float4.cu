#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include "generator.h"

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

const int M = 1024;
const int N = 1024;
const int K = 1024;
const int BM = 128;
const int BN = 128;
const int BK = 8;
const int TM = 8;
const int TN = 8;

void cudaCheck(hipError_t error) {
    if (error != hipSuccess) {
        exit(EXIT_FAILURE);
    }
    return;
};

void cpuSgemm(
    float *a, float *b, float *c, const int M, const int N, const int K) {
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = psum;
        }
    }
}

__global__ void naiveSgemm(int M, int N, int K, float *A, float *B,  float *C) {

  const int threadRow = threadIdx.x / (BN / TN);
  const int threadCol = threadIdx.x % (BN / TN);

  __shared__ float sharedA[BM * BK];
  __shared__ float sharedB[BK * BN];

  float resultPerThread[TM * TN] = {0.0};
  float vectorOuterA[TM] = {0.0};
  float vectorOuterB[TN] = {0.0};

  A += blockIdx.y * BM * K;
  B += blockIdx.x * BN;
  C += blockIdx.y * BM * N + blockIdx.x * BM;

  const int sharedARow = threadIdx.x / (BK / 4);
  const int sharedACol = threadIdx.x % (BK / 4);
  const int sharedBRow = threadIdx.x / (BN / 4);
  const int sharedBCol = threadIdx.x % (BN / 4);

  for (int dotOrder = 0; dotOrder < K; dotOrder += BK) {
    reinterpret_cast<float4 *>(&sharedA[sharedARow * BK + sharedACol * 4])[0] = reinterpret_cast<float4 *>(&A[sharedARow * K + sharedACol * 4])[0];
    reinterpret_cast<float4 *>(&sharedB[sharedBRow * BN + sharedBCol * 4])[0] = reinterpret_cast<float4 *>(&B[sharedBRow * N + sharedBCol * 4])[0];

    __syncthreads();

    A += BK;
    B += BK * N;

    for (int innerOuterProdOrder = 0; innerOuterProdOrder < BK;
         innerOuterProdOrder++) {
        for (int i = 0; i < TM; i++) {
            vectorOuterA[i] =
            sharedA[(threadRow * TM + i) * BK + innerOuterProdOrder];
        }
        for (int i = 0; i < TN; i++) {
            vectorOuterB[i] =
            sharedB[innerOuterProdOrder * BN + (threadCol * TN + i)];
        }

        for (int resultRow = 0; resultRow < TM; resultRow++) {
            for (int resultCol = 0; resultCol < TN; resultCol++) {
                resultPerThread[resultRow * TN + resultCol] +=
              vectorOuterA[resultRow] * vectorOuterB[resultCol];
            }
        }
        

    }

    __syncthreads();

  }

  for (int resultRow = 0; resultRow < TM; resultRow += 1) {
    for (int resultCol = 0; resultCol < TN; resultCol += 4) { 
        float4 tmp;

        tmp.x = resultPerThread[resultRow * TN + resultCol + 0];
      tmp.y = resultPerThread[resultRow * TN + resultCol + 1];
      tmp.z = resultPerThread[resultRow * TN + resultCol + 2];
      tmp.w = resultPerThread[resultRow * TN + resultCol + 3];

      reinterpret_cast<float4 *>(&C[(threadRow * TM + resultRow) * N +
                                    (threadCol * TN + resultCol)])[0] = tmp;
    }
  }

}


float testMaxError(
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a = new float[M * K];
    float *h_b = new float[K * N];
    float *h_c = new float[M * N];
    float *h_d_c = new float[M * N];
    float *d_a, *d_b, *d_c;

    generate_array(h_a, M * K);
    generate_array(h_b, K * N);
    generate_array(h_c, M * N);
    generate_array(h_d_c, M * N);

    cudaCheck(hipMalloc(&d_a, size_a));
    cudaCheck(hipMalloc(&d_b, size_b));
    cudaCheck(hipMalloc(&d_c, size_c));

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    
    cudaCheck(hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice));
    printf("Before kernel launch\n");
    naiveSgemm<<<gridDim, blockDim>>>( M, N, K, d_a, d_b, d_c);
    cudaCheck(hipDeviceSynchronize());
    printf("After kernel launch\n");

    cudaCheck(hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost));

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs(h_d_c[i] - h_c[i]);
        // if (this_error > 0) {
        //     printf("index %d, host calculate %f, device %f, error %f\n", i, h_c[i], h_d_c[i], this_error);
        // }
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}

int main() {

    printf("\nKernal = sgemm thread tailing\n");

    {
        int thread_num = BM * BN / TM / TN;
        dim3 blockDim(thread_num);
        dim3 gridDim((M - 1) / BM + 1, (N - 1) / BN + 1);
        float max_error = testMaxError(gridDim, blockDim, M, N, K);
        printf("Max Error = %f\n", max_error);
    }

    return 0;
}